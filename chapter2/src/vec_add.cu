
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <sys/time.h>

using namespace std;

void vecAdd(float* A, float* B, float* C, int n) {
    for (int i = 0; i < n; i++) {
        C[i] = A[i] + B[i];
    }
}

__global__
void vecAddKernel(float* A_d, float* B_d, float* C_d, int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) C_d[i] = A_d[i] + B_d[i];
}


void cpu_add(int n) {
    cout << "cpu add: " << n << endl;

    size_t size = n * sizeof(float);

    // host memery
    float *a = (float *)malloc(size);
    float *b = (float *)malloc(size);
    float *c = (float *)malloc(size);

    for (int i = 0; i < n; i++) {
        float af = rand() / double(RAND_MAX);
        float bf = rand() / double(RAND_MAX);
        a[i] = af;
        b[i] = bf;
        c[i] = a[i] + b[i];
    }
    printf("add one.\n");

   struct timeval t1, t2;

    gettimeofday(&t1, NULL);

    vecAdd(a, b, c, n);

    gettimeofday(&t2, NULL);

    //for (int i = 0; i < 10; i++) 
    //    cout << vecA[i] << " " << vecB[i] << " " << vecC[i] << endl;
    double timeuse = (t2.tv_sec - t1.tv_sec) + (double)(t2.tv_usec - t1.tv_usec)/1000000.0;
    cout  << timeuse << endl;

    free(a);
    free(b);
    free(c);
}


void gpu_add(int n) {
    cout  << "gpu add: " << n << endl;

    size_t size = n * sizeof(float);

    // host memery
    float *a = (float *)malloc(size);
    float *b = (float *)malloc(size);
    float *c = (float *)malloc(size);

    for (int i = 0; i < n; i++) {
        float af = rand() / double(RAND_MAX);
        float bf = rand() / double(RAND_MAX);
        a[i] = af;
        b[i] = bf;
    }

    float *da = NULL;
    float *db = NULL;
    float *dc = NULL;

    hipMalloc((void **)&da, size);
    hipMalloc((void **)&db, size);
    hipMalloc((void **)&dc, size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    struct timeval t1, t2;

    int threadPerBlock = 256;
    int blockPerGrid = (n + threadPerBlock - 1)/threadPerBlock;
    printf("threadPerBlock: %d \nblockPerGrid: %d \n",threadPerBlock,blockPerGrid);

    gettimeofday(&t1, NULL);

    vecAddKernel <<< blockPerGrid, threadPerBlock >>> (da, db, dc, n);

    gettimeofday(&t2, NULL);

    hipMemcpy(c,dc,size,hipMemcpyDeviceToHost);

    //for (int i = 0; i < 10; i++) 
    //    cout << vecA[i] << " " << vecB[i] << " " << vecC[i] << endl;
    double timeuse = (t2.tv_sec - t1.tv_sec) + (double)(t2.tv_usec - t1.tv_usec)/1000000.0;
    cout << timeuse << endl;

    hipFree(da);
    hipFree(db);
    hipFree(dc);

    free(a);
    free(b);
    free(c);
}


int main(int argc, char *argv[]) {

    cout << "argc: " << argc << endl;
    if (argc < 2) {
        return 0;
    }
    int n = atoi(argv[1]);
    cpu_add(n);
    gpu_add(n);
    return 0;

}